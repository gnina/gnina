#include "hip/hip_runtime.h"
/*
 model.cu

 Contains the GPU-specific methods of model.
 Currently also have all eval methods (cpu or gpu) here for easy reference.

 */

#include "model.h"
#include "common.h"
#include "file.h"
#include "curl.h"
#include <boost/unordered_map.hpp>
#include <boost/algorithm/string/predicate.hpp>
#include "non_cache_gpu.h"
#include "loop_timer.h"
#include "gpu_debug.h"
#include "device_buffer.h"

#define MAX_THREADS 1024

fl model::eval_interacting_pairs(const precalculate& p, fl v,
    const interacting_pairs& pairs, const vecv& coords) const { // clean up
  const fl cutoff_sqr = p.cutoff_sqr();
  fl e = 0;
  VINA_FOR_IN(i, pairs) {
    const interacting_pair& ip = pairs[i];
    fl r2 = vec_distance_sqr(coords[ip.a], coords[ip.b]);
    if (r2 < cutoff_sqr) {
      fl tmp = p.eval(atoms[ip.a], atoms[ip.b], r2);
      curl(tmp, v);
      e += tmp;
    }
  }
  return e;
}

fl model::eval_interacting_pairs_deriv(const precalculate& p, fl v,
    const interacting_pairs& pairs, const vecv& coords, vecv& forces) const { // adds to forces  // clean up
  const fl cutoff_sqr = p.cutoff_sqr();
  fl e = 0;
  VINA_FOR_IN(i, pairs) {
    const interacting_pair& ip = pairs[i];
    vec r;
    r = coords[ip.b] - coords[ip.a]; // a -> b
    fl r2 = sqr(r);
    if (r2 < cutoff_sqr) {
      pr tmp = p.eval_deriv(atoms[ip.a], atoms[ip.b], r2);
      vec force;
      force = tmp.second * r;
      curl(tmp.first, force, v);
      e += tmp.first;

      // FIXME inefficient, if using hard curl
      forces[ip.a] -= force; // we could omit forces on inflex here
      forces[ip.b] += force;
    }
  }
  return e;
}

//evaluates interacting pairs (which is all of them) on the gpu
template<typename infoT>
__host__  __device__ fl gpu_data::eval_interacting_pairs_deriv_gpu(
    const infoT& info, fl v, interacting_pair* pairs, unsigned pairs_sz) const { // adds to forces  // clean up

  float e = 0;
  // If there aren't any pairs, just return.
  if (pairs_sz == 0) {
    return e;
  }

  const fl cutoff_sqr = info.cutoff_sq;
#ifdef __CUDA_ARCH__
  scratch[0] = 0;
#else
  hipMemsetAsync(scratch, 0, sizeof(float), hipStreamPerThread);
#endif

  //TODO: this should not be a dynamic launch...
  if (pairs_sz < CUDA_THREADS_PER_BLOCK) {
    eval_intra_kernel<<<1, pairs_sz>>>(info.splineInfo, coords, pairs, pairs_sz,
        cutoff_sqr, v, minus_forces, scratch);

  } else {
    eval_intra_kernel<<<CUDA_GET_BLOCKS(pairs_sz, 1024),
    CUDA_THREADS_PER_BLOCK>>>(info.splineInfo, coords, pairs, pairs_sz,
        cutoff_sqr, v, minus_forces, scratch);
  }
#ifdef __CUDA_ARCH__ 
  //hipDeviceSynchronize();
  return scratch[0];
#else
  float out;
  definitelyPinnedMemcpy(&out, scratch, sizeof(float), hipMemcpyDeviceToHost);
  return out;
#endif
}

fl model::evali(const precalculate& p, const vec& v){ // clean up

  assert(0);
  /* TODO */
  /* fl e = 0; */
  /* VINA_FOR_IN(i, ligands) */
  /* 	e += eval_interacting_pairs(p, v[0], ligands[i].pairs, internal_coords); */
  //probably might as well use coords here
  /* return e; */
  return 0;
}

fl model::evale(const precalculate& p, const igrid& ig, const vec& v) { // clean up
  fl e = ig.eval(*this, v[1]);
  e += eval_interacting_pairs(p, v[2], other_pairs, coords);
  return e;
}

fl model::eval(const precalculate& p, const igrid& ig, const vec& v,
    const conf& c, const grid& user_grid) { // clean up
  set(c);
  fl e = evale(p, ig, v);
  VINA_FOR_IN(i, ligands)
    e += eval_interacting_pairs(p, v[0], ligands[i].pairs, coords); // coords instead of internal coords
  //std::cout << "smina_contribution: " << e << "\n";
  if (user_grid.initialized()) {
    fl uge = 0;
    vecv l_coords = this->get_ligand_coords();
    VINA_FOR_IN(i, l_coords) {
      fl tmp = user_grid.evaluate_user(l_coords[i], (fl) 1000);
      uge += tmp;
      e += tmp;
    }
    std::cout << "User_grid_contribution: " << uge << "\n";
  }

  return e;
}

static __global__
void derivatives_kernel(tree_gpu *t, const vec * coords, const vec* forces,
    change_gpu c) {
  t->derivative(coords, forces, &c);
}

static __global__
void set_conf_kernel(tree_gpu *t, const vec *atom_coords, vec *coords,
    const conf_gpu c) {
  t->set_conf(atom_coords, coords, &c);
}

template<typename infoT>
__device__ fl gpu_data::eval_deriv_gpu(const infoT& info, const vec& v,
    const conf_gpu& c, change_gpu& g) {
  // static loop_timer t;
  // t.resume();
  fl e, ie = 0;
  if (threadIdx.x == 0) {
    set_conf_kernel<<<1, treegpu->num_atoms>>>(treegpu, atom_coords,
        (vec*) coords, c);
    memset(minus_forces, 0, sizeof(force_energy_tup) * (forces_size));
    //hipDeviceSynchronize();
    assert(0); //TODO: refactor to not require synchronization
    e = single_point_calc(info, coords, minus_forces, v[1]);
    //hipDeviceSynchronize();
    if (other_pairs_size)
      e += eval_interacting_pairs_deriv_gpu(info, v[2], other_pairs,
          other_pairs_size);
    if (pairs_size)
      ie = eval_interacting_pairs_deriv_gpu(info, v[0], interacting_pairs,
          pairs_size); // adds to minus_forces

    e += ie;
    derivatives_kernel<<<1, treegpu->num_atoms>>>(treegpu, (vec*) coords,
        (vec*) minus_forces, g);

    //hipDeviceSynchronize();
  }

  // t.stop();

  /* flex.derivative(coords, minus_forces, g.flex); // inflex forces are ignored */
  return e;
}

//NB: the next two are only used for score_only, but they also compute the forces
//unnecessarily. this is fine as long as you don't care about using score_only
//for performance metrics, just correctness, which is what you should do. 
fl gpu_data::eval(const GPUNonCacheInfo& info, const float v) {
  fl e;
  hipMemsetAsync(minus_forces, 0,
      sizeof(force_energy_tup) * info.num_movable_atoms, hipStreamPerThread);
  e = single_point_calc(info, coords, minus_forces, v);
  return e;
}

fl gpu_data::eval_intramolecular(const GPUNonCacheInfo& info, const float v) {
  fl ie;
  ie = eval_interacting_pairs_deriv_gpu(info, v, interacting_pairs, pairs_size);
  return ie;
}

fl model::eval_deriv(const precalculate& p, const igrid& ig, const vec& v,
    const conf& c, change& g, const grid& user_grid) { // clean up
  set(c);

  fl e = ig.eval_deriv(*this, v[1], user_grid); // sets minus_forces, except inflex
  fl ie = 0;

  if (!ig.skip_interacting_pairs()) {
    ie += eval_interacting_pairs_deriv(p, v[2], other_pairs, coords,
        minus_forces); // adds to minus_forces

    VINA_FOR_IN(i, ligands)
      ie += eval_interacting_pairs_deriv(p, v[0], ligands[i].pairs, coords,
          minus_forces); // adds to minus_forces
    e += ie;
  }

  // calculate derivatives
  ligands.derivative(coords, minus_forces, g.ligands);
  flex.derivative(coords, minus_forces, g.flex); // inflex forces are ignored
  g.receptor = rec_change; //for cnn

  return e;
}

fl model::eval_intra(const precalculate& p, const vec& v) {
  fl ie = 0;
  VINA_FOR_IN(i, ligands)
    ie += eval_interacting_pairs_deriv(p, v[0], ligands[i].pairs, coords,
        minus_forces); // adds to minus_forces
  return ie;
}

void model::clear_minus_forces() {
  minus_forces.clear();
  minus_forces.reserve(m_num_movable_atoms);
  VINA_FOR(i, m_num_movable_atoms) {
    vec force;
    force.data[0] = 0.0;
    force.data[1] = 0.0;
    force.data[2] = 0.0;
    minus_forces.push_back(force);
  }
}

void model::add_minus_forces(const std::vector<gfloat3>& forces) {
  assert(forces.size() <= m_num_movable_atoms);
  unsigned j = 0;
  VINA_FOR(i, m_num_movable_atoms) {
    if (!atoms[i].is_hydrogen()) // no hydrogen forces
    {
      minus_forces[i].data[0] += forces[j].x;
      minus_forces[i].data[1] += forces[j].y;
      minus_forces[i].data[2] += forces[j].z;
      j += 1;
    }
  }
}

void model::sub_minus_forces(const std::vector<gfloat3>& forces) {
  assert(forces.size() <= m_num_movable_atoms);
  unsigned j = 0;
  VINA_FOR(i, m_num_movable_atoms) {
    if (!atoms[i].is_hydrogen()) // no hydrogen forces
    {
      minus_forces[i].data[0] -= forces[j].x;
      minus_forces[i].data[1] -= forces[j].y;
      minus_forces[i].data[2] -= forces[j].z;
      j += 1;
    }
  }
}

void model::scale_minus_forces(fl scale) {
  unsigned j = 0;
  VINA_FOR(i, m_num_movable_atoms) {
    if (!atoms[i].is_hydrogen()) // no hydrogen forces
    {
      minus_forces[i].data[0] *= scale;
      minus_forces[i].data[1] *= scale;
      minus_forces[i].data[2] *= scale;
      j += 1;
    }
  }
}

/* This is my attempt to make gpus more deterministic.. */
void model::round_minus_forces() {
  VINA_FOR(i, m_num_movable_atoms) {
    if (!atoms[i].is_hydrogen()) // no hydrogen forces
    {
      for(unsigned j = 0; j < 3; j++) {
          double val = minus_forces[i].data[j];
          val *= 10000; 
          val = roundf(val);
          minus_forces[i].data[j] = val/10000;
      }
    }
  }
}

fl model::get_minus_forces_sum_magnitude() const {
  fl x = 0, y = 0, z = 0;
  VINA_FOR(i, m_num_movable_atoms) {
    if (!atoms[i].is_hydrogen()) // no hydrogen forces
    {
      x += minus_forces[i].data[0];
      y += minus_forces[i].data[1];
      z += minus_forces[i].data[2];
    }
  }
  return sqrt(x * x + y * y + z * z);
}

//evaluate interactiongs between all of flex (including rigid) and protein
//will ignore grid_atoms greater than max
fl model::eval_flex(const precalculate& p, const vec& v, const conf& c,
    unsigned maxGridAtom) {
  set(c);
  fl e = 0;
  sz nat = num_atom_types();
  const fl cutoff_sqr = p.cutoff_sqr();

  //ignore atoms after maxGridAtom (presumably part of "unfrag")
  sz gridstop = grid_atoms.size();
  if (maxGridAtom > 0 && maxGridAtom < gridstop) gridstop = maxGridAtom;

  // flex-rigid
  VINA_FOR(i, atoms.size()) {
    if (find_ligand(i) < ligands.size()) continue; // we only want flex-rigid interaction
    const atom& a = atoms[i];
    smt t1 = a.get();
    if (t1 >= nat) continue;
    VINA_FOR_IN(j, grid_atoms) {
      if (j >= gridstop) break;
      const atom& b = grid_atoms[j];
      smt t2 = b.get();
      if (t2 >= nat) continue;
      fl r2 = vec_distance_sqr(coords[i], b.coords);
      if (r2 < cutoff_sqr) {
        fl this_e = p.eval(a, b, r2);
        curl(this_e, v[1]);
        e += this_e;
      }
    }
  }

  return e;
}

fl model::eval_intramolecular(const precalculate& p, const vec& v,
    const conf& c) {
  set(c);
  fl e = 0;

  // internal for each ligand
  VINA_FOR_IN(i, ligands)
    e += eval_interacting_pairs(p, v[0], ligands[i].pairs, coords); // coords instead of internal coords

  sz nat = num_atom_types();
  const fl cutoff_sqr = p.cutoff_sqr();

  // flex-rigid
  VINA_FOR(i, num_movable_atoms()) {
    if (find_ligand(i) < ligands.size()) continue; // we only want flex-rigid interaction
    const atom& a = atoms[i];
    if(a.iscov) continue; //covalent ligand treated as ligand
    smt t1 = a.get();
    if (t1 >= nat || is_hydrogen(t1)) continue;
    VINA_FOR_IN(j, grid_atoms) {
      const atom& b = grid_atoms[j];
      if(b.iscov) continue;
      smt t2 = b.get();
      if (t2 >= nat || is_hydrogen(t2)) continue;
      fl r2 = vec_distance_sqr(coords[i], b.coords);
      if (r2 < cutoff_sqr) {
        fl this_e = p.eval(a, b, r2);
        curl(this_e, v[1]);
        e += this_e;
      }
    }
  }

// flex-flex
  VINA_FOR_IN(i, other_pairs) {
    const interacting_pair& pair = other_pairs[i];
    if (find_ligand(pair.a) < ligands.size()
        || find_ligand(pair.b) < ligands.size()) continue; // we only need flex-flex
    if(atoms[pair.a].iscov || atoms[pair.b].iscov) continue;
    fl r2 = vec_distance_sqr(coords[pair.a], coords[pair.b]);
    if (r2 < cutoff_sqr) {
      fl this_e = p.eval(atoms[pair.a], atoms[pair.b], r2);
      curl(this_e, v[2]);
      e += this_e;
    }
  }
  return e;
}

fl model::eval_adjusted(const scoring_function& sf, const precalculate& p,
    const igrid& ig, const vec& v, const conf& c, fl intramolecular_energy,
    const grid& user_grid) {
  fl e = eval(p, ig, v, c, user_grid); // sets c
  return sf.conf_independent(*this, e - intramolecular_energy);
}

void model::initialize_gpu() {
  //TODO: only re-malloc if need larger size
  deallocate_gpu();

  CUDA_CHECK_GNINA(device_malloc(&gdata.coords, sizeof(vec) * coords.size()));
  CUDA_CHECK_GNINA(
      device_malloc(&gdata.atom_coords, sizeof(vec) * atoms.size()));
  CUDA_CHECK_GNINA(
      device_malloc(&gdata.minus_forces, sizeof(vec) * minus_forces.size()));
  CUDA_CHECK_GNINA(device_malloc(&gdata.scratch, sizeof(float)));

  gdata.coords_size = coords.size();
  gdata.atom_coords_size = atoms.size();
  gdata.forces_size = minus_forces.size();

  //ligand internal pairs 
  if (ligands.size()) {
    std::vector<interacting_pair> ligand_pairs(ligands[0].pairs);
    for (int i = 1; i < ligands.size(); i++)
      ligand_pairs.insert(ligand_pairs.end(), ligands[i].pairs.begin(),
          ligands[i].pairs.end());
    gdata.pairs_size = ligand_pairs.size();

    CUDA_CHECK_GNINA(
        device_malloc(&gdata.interacting_pairs,
            sizeof(interacting_pair) * ligand_pairs.size()));
    CUDA_CHECK_GNINA(
        definitelyPinnedMemcpy(gdata.interacting_pairs, &ligand_pairs[0],
            sizeof(interacting_pair) * ligand_pairs.size(),
            hipMemcpyHostToDevice));
  }

  //all flexible pairs, but not intra
  if (other_pairs.size()) {
    CUDA_CHECK_GNINA(
        device_malloc(&gdata.other_pairs,
            sizeof(interacting_pair) * other_pairs.size()));
    CUDA_CHECK_GNINA(
        definitelyPinnedMemcpy(gdata.other_pairs, &other_pairs[0],
            sizeof(interacting_pair) * other_pairs.size(),
            hipMemcpyHostToDevice));
    gdata.other_pairs_size = other_pairs.size();
  }

  //input atom coords do not change
  std::vector<vec> acoords(atoms.size());
  for (unsigned i = 0, n = atoms.size(); i < n; i++) {
    acoords[i] = atoms[i].coords;
  }

  //set up tree. Writes padding to mark every atom in acoords with its owner.
  //TODO: quite intrusive
  tree_gpu tg(ligands, flex, &acoords[0], gdata.dfs_order_bfs_indices,
      gdata.bfs_order_dfs_indices);
  gdata.nlig_roots = tg.nlig_roots;
  CUDA_CHECK_GNINA(device_malloc(&gdata.treegpu, sizeof(tree_gpu)));
  CUDA_CHECK_GNINA(
      definitelyPinnedMemcpy(gdata.treegpu, &tg, sizeof(tree_gpu),
          hipMemcpyHostToDevice));

  //this contains the marked_coords for all the atoms and therefore all the
  //nodes in both trees
  CUDA_CHECK_GNINA(
      definitelyPinnedMemcpy(gdata.atom_coords, &acoords[0],
          sizeof(vec) * atoms.size(), hipMemcpyHostToDevice));

  CUDA_CHECK_GNINA(
      definitelyPinnedMemcpy(gdata.coords, &coords[0],
          coords.size() * sizeof(atom_params), hipMemcpyHostToDevice));
}

void model::deallocate_gpu() {
  if (gdata.coords) {
    CUDA_CHECK_GNINA(device_free(gdata.coords));
    gdata.coords = NULL;
  }
  if (gdata.atom_coords) {
    CUDA_CHECK_GNINA(device_free(gdata.atom_coords));
    gdata.atom_coords = NULL;
  }
  if (gdata.minus_forces) {
    CUDA_CHECK_GNINA(device_free(gdata.minus_forces));
    gdata.minus_forces = NULL;
  }
  if (gdata.treegpu) {
    tree_gpu::deallocate(gdata.treegpu);
    gdata.treegpu = NULL;
  }
  if (gdata.scratch) {
    CUDA_CHECK_GNINA(device_free(gdata.scratch));
    gdata.scratch = NULL;
  }
  if (gdata.dfs_order_bfs_indices) {
    delete[] gdata.dfs_order_bfs_indices;
    gdata.dfs_order_bfs_indices = NULL;
  }
  if (gdata.bfs_order_dfs_indices) {
    delete[] gdata.bfs_order_dfs_indices;
    gdata.bfs_order_dfs_indices = NULL;
  }

  if (gdata.interacting_pairs) {
    CUDA_CHECK_GNINA(device_free(gdata.interacting_pairs));
    gdata.interacting_pairs = NULL;
  }
  gdata.coords_size = gdata.atom_coords_size = gdata.forces_size =
      gdata.pairs_size = 0;
}

//copy relevant data to gpu buffers
void gpu_data::copy_to_gpu(model& m) {
  CUDA_CHECK_GNINA(
      definitelyPinnedMemcpy(coords, &m.coords[0], coords_size * sizeof(vec),
          hipMemcpyHostToDevice));

  //minus_forces gets initialized in eval_deriv
  //interacting pairs, atom_coords and ligand tree do not change
}

//copy back relevant data from gpu buffers
void gpu_data::copy_from_gpu(model& m) {
  assert(coords);
  CUDA_CHECK_GNINA(
      definitelyPinnedMemcpy(&m.coords[0], coords, coords_size * sizeof(vec),
          hipMemcpyDeviceToHost));
}

size_t gpu_data::node_idx_cpu2gpu(size_t cpu_idx) const {
  return dfs_order_bfs_indices[cpu_idx];
}

template
__host__  __device__ fl gpu_data::eval_interacting_pairs_deriv_gpu<
    GPUNonCacheInfo>(const GPUNonCacheInfo& info, fl v, interacting_pair* pairs,
    unsigned pairs_sz) const;

template
__host__  __device__ fl gpu_data::eval_interacting_pairs_deriv_gpu<GPUCacheInfo>(
    const GPUCacheInfo& info, fl v, interacting_pair* pairs,
    unsigned pairs_sz) const;

template __device__ fl gpu_data::eval_deriv_gpu<GPUNonCacheInfo>(
    const GPUNonCacheInfo& info, const vec& v, const conf_gpu& c,
    change_gpu& g);
template __device__ fl gpu_data::eval_deriv_gpu<GPUCacheInfo>(
    const GPUCacheInfo& info, const vec& v, const conf_gpu& c, change_gpu& g);


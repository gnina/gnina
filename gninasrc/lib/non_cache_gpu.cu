#include "non_cache_gpu.h"
#include "loop_timer.h"
#include "gpu_math.h"
#include "device_buffer.h"

non_cache_gpu::non_cache_gpu(szv_grid_cache& gcache, const grid_dims& gd_,
    const precalculate_gpu* p_, fl slope_)
    : non_cache(gcache, gd_, p_, slope_) {
  const model& m = gcache.getModel();
  info.cutoff_sq = p->cutoff_sqr();
  info.slope = slope;

  unsigned num_movable_atoms = m.num_movable_atoms();
  info.num_movable_atoms = num_movable_atoms;
  //allocate memory for positions, partial charges, and atom types of movable atoms
  //TODO: remove penalties? I think this is never being used
  thread_buffer.alloc(&info.lig_penalties,
      sizeof(force_energy_tup[num_movable_atoms]));
  thread_buffer.alloc(&info.types, sizeof(unsigned[num_movable_atoms]));

  //initialize atom types and partial charges
  std::vector<unsigned> htypes(num_movable_atoms);

  VINA_FOR(i, num_movable_atoms) {
    htypes[i] = m.atoms[i].get();
    /* TODO breaking const */
    ((atom_params *) &m.coords[0])[i].charge = m.atoms[i].charge;
    /* lig_atoms_scratch[i].charge = 101010; */
  }
  definitelyPinnedMemcpy(info.types, &htypes[0],
      sizeof(unsigned[num_movable_atoms]), hipMemcpyHostToDevice);

  info.gridbegins = gfloat3(gd[0].begin, gd[1].begin, gd[2].begin);
  info.gridends = gfloat3(gd[0].end, gd[1].end, gd[2].end);

  //figure out all possibly relevant receptor atoms
  szv recatomids;
  gcache.compute_relevant(gd_, recatomids);
  unsigned nrec_atoms = recatomids.size();
  info.nrec_atoms = nrec_atoms;

  //allocate memory for positions, atom types, and partial charges of all
  //possibly relevant receptor atoms
  thread_buffer.alloc(&info.rec_atoms, sizeof(atom_params[nrec_atoms]));
  thread_buffer.alloc(&info.rectypes, sizeof(unsigned[nrec_atoms]));

  //initialize
  std::vector<atom_params> hrec_atoms(nrec_atoms);
  std::vector<unsigned> hrectypes(nrec_atoms);
  for (unsigned i = 0; i < nrec_atoms; i++) {
    unsigned index = recatomids[i];
    const vec& c = m.grid_atoms[index].coords;
    atom_params *a = &hrec_atoms[i];
    a->coords.x = c[0];
    a->coords.y = c[1];
    a->coords.z = c[2];
    a->charge = m.grid_atoms[index].charge;

    hrectypes[i] = m.grid_atoms[index].get();
  }
  definitelyPinnedMemcpy(info.rec_atoms, &hrec_atoms[0],
      sizeof(atom_params[nrec_atoms]), hipMemcpyHostToDevice);
  definitelyPinnedMemcpy(info.rectypes, &hrectypes[0],
      sizeof(unsigned[nrec_atoms]), hipMemcpyHostToDevice);

  info.ntypes = p_->num_types();
  info.splineInfo = p_->getDeviceData();
}

non_cache_gpu::~non_cache_gpu() {
  //deallocate device memory
  thread_buffer.reinitialize();
}

fl non_cache_gpu::eval(model& m, fl v) const {
  abort(); //not implemented
}

void non_cache_gpu::setSlope(fl sl) {
  slope = sl;
  info.slope = sl;
}

